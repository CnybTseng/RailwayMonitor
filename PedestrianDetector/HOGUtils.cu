#include "hip/hip_runtime.h"
#include "HOGUtils.h"

//Round a / b to nearest higher integer value
__host__ int iDivUp(int a, int b) { return (a % b != 0) ? (a / b + 1) : (a / b); }

//Round a / b to nearest lower integer value
__host__ int iDivDown(int a, int b) { return a / b; }

//Align a to nearest higher multiple of b
__host__ int iAlignUp(int a, int b) { return (a % b != 0) ?  (a - a % b + b) : a; }

//Align a to nearest lower multiple of b
__host__ int iAlignDown(int a, int b)  {return a - a % b; }

//Round a / b to nearest higher integer value
__host__ int iDivUpF(int a, float b) { return (a % int(b) != 0) ? int(a / b + 1) : int(a / b);}

__host__ int iClosestPowerOfTwo(int x) { x--; x |= x >> 1; x |= x >> 2; x |= x >> 4; x |= x >> 8; x |= x >> 16; x++; return x; }

__host__ void Uchar4ToFloat4(uchar4 *inputImage, float4 *outputImage, int width, int height)
{
	dim3 threads_in_block(16,16);
	dim3 blocks(iDivUp(width,16), iDivUp(height,16));
	uchar4tofloat4<<<blocks, threads_in_block>>>(inputImage, outputImage, width, height);
}
__host__ void Float4ToUchar4(float4 *inputImage, uchar4 *outputImage, int width, int height)
{
	dim3 threads_in_block(16,16);
	dim3 blocks(iDivUp(width,16), iDivUp(height,16));
	float4toUchar4<<<blocks, threads_in_block>>>(inputImage, outputImage, width, height);
}
__host__ void Float2ToUchar4(float2 *inputImage, uchar4 *outputImage, int width, int height, int index)
{
	dim3 threads_in_block(16,16);
	dim3 blocks(iDivUp(width,16), iDivUp(height,16));
	float2toUchar4<<<blocks, threads_in_block>>>(inputImage, outputImage, width, height, index);
}
__host__ void Float2ToUchar1(float2 *inputImage, uchar1 *outputImage, int width, int height, int index)
{
	dim3 threads_in_block(16,16);
	dim3 blocks(iDivUp(width,16), iDivUp(height,16));
	float2toUchar1<<<blocks, threads_in_block>>>(inputImage, outputImage, width, height, index);
}
__host__ void Float1ToUchar4(float1 *inputImage, uchar4 *outputImage, int width, int height)
{
	dim3 threads_in_block(16,16);
	dim3 blocks(iDivUp(width,16), iDivUp(height,16));
	float1toUchar4<<<blocks, threads_in_block>>>(inputImage, outputImage, width, height);
}
__host__ void Float1ToUchar1(float1 *inputImage, uchar1 *outputImage, int width, int height)
{
	dim3 threads_in_block(16,16);
	dim3 blocks(iDivUp(width,16), iDivUp(height,16));
	float1toUchar1<<<blocks, threads_in_block>>>(inputImage, outputImage, width, height);
}
__global__ void float4toUchar4(float4 *inputImage, uchar4 *outputImage, int width, int height)
{
	int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
	int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

	float4 pixelf = inputImage[offset];
	uchar4 pixel;
	pixel.x = (unsigned char) pixelf.x; pixel.y = (unsigned char) pixelf.y;
	pixel.z = (unsigned char) pixelf.z; pixel.w = (unsigned char) pixelf.w;

	outputImage[offset] = pixel;
}
__global__ void float2toUchar4(float2 *inputImage, uchar4 *outputImage, int width, int height, int index)
{
	int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
	int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

	float2 pixelf = inputImage[offset];
	float pixelfIndexed = (index == 0) ? pixelf.x : pixelf.y;
	
	// uchar4 pixel;
	// pixel.x = (unsigned char) abs(pixelfIndexed); pixel.y = (unsigned char) abs(pixelfIndexed);
	// pixel.z = (unsigned char) abs(pixelfIndexed); pixel.w = (unsigned char) abs(pixelfIndexed);
	
	// outputImage[offset] = pixel;
	memcpy(outputImage + offset, &pixelfIndexed, sizeof(float));
}
__global__ void float2toUchar1(float2 *inputImage, uchar1 *outputImage, int width, int height, int index)
{
	int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
	int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

	float2 pixelf = inputImage[offset];
	float pixelfIndexed = (index == 0) ? pixelf.x : pixelf.y;

	uchar1 pixel;
	pixel.x = (unsigned char) pixelfIndexed;

	outputImage[offset] = pixel;
}
__global__ void float1toUchar4(float1 *inputImage, uchar4 *outputImage, int width, int height)
{
	int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
	int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

	float1 pixelf = inputImage[offset];
	uchar4 pixel;
	pixel.x = (unsigned char) pixelf.x; pixel.y = (unsigned char) pixelf.x;
	pixel.z = (unsigned char) pixelf.x; pixel.w = (unsigned char) pixelf.x;

	outputImage[offset] = pixel;
}
__global__ void float1toUchar1(float1 *inputImage, uchar1 *outputImage, int width, int height)
{
	int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
	int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

	float1 pixelf = inputImage[offset];
	uchar1 pixel;
	pixel.x = (unsigned char) pixelf.x;

	outputImage[offset] = pixel;
}

__global__ void uchar4tofloat4(uchar4 *inputImage, float4 *outputImage, int width, int height)
{
	int offsetX = blockIdx.x * blockDim.x + threadIdx.x;
	int offsetY = blockIdx.y * blockDim.y + threadIdx.y;

	if (offsetX < width && offsetY < height)
	{
		int offsetBlock = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y * width;
		int offset = offsetBlock + threadIdx.x + threadIdx.y * width;

		uchar4 pixel = inputImage[offset];
		float4 pixelf;
		pixelf.x = pixel.x; pixelf.y = pixel.y;
		pixelf.z = pixel.z; pixelf.w = pixel.w;

		outputImage[offset] = pixelf;
	}
}
