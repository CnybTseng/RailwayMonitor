#include "hip/hip_runtime.h"
#include "HOGPadding.h"
#include "HOGUtils.h"

extern int hWidthROI, hHeightROI;
extern int hPaddedWidth, hPaddedHeight;
extern int hWidth, hHeight;
extern int hPaddingSizeX, hPaddingSizeY;

extern int avSizeX, avSizeY, marginX, marginY;

uchar4* paddedRegisteredImageU4;

__host__ void InitPadding(int hPaddedWidth, int hPaddedHeight)
{
	checkCudaErrors(hipMalloc((void**) &paddedRegisteredImageU4, sizeof(uchar4) * hPaddedWidth * hPaddedHeight));
}

__host__ void ClosePadding()
{
	checkCudaErrors(hipFree(paddedRegisteredImageU4));
}

__host__ void PadHostImage(uchar4* registeredImage, float4 *paddedRegisteredImage,
		int minx, int miny, int maxx, int maxy)
{
	hWidthROI = maxx - minx;
	hHeightROI = maxy - miny;

	int toaddxx = 0, toaddxy = 0, toaddyx = 0, toaddyy = 0;

	if (avSizeX) { toaddxx = hWidthROI * marginX / avSizeX; toaddxy = hHeightROI * marginY / avSizeX; }
	if (avSizeY) { toaddyx = hWidthROI * marginX / avSizeY; toaddyy = hHeightROI * marginY / avSizeY; }

	hPaddingSizeX = max(toaddxx, toaddyx); hPaddingSizeY = max(toaddxy, toaddyy);

	hPaddedWidth = hWidthROI + hPaddingSizeX*2;
	hPaddedHeight = hHeightROI + hPaddingSizeY*2;

	checkCudaErrors(hipMemset(paddedRegisteredImageU4, 0, sizeof(uchar4) * hPaddedWidth * hPaddedHeight));

	checkCudaErrors(hipMemcpy2D(paddedRegisteredImageU4 + hPaddingSizeX + hPaddingSizeY * hPaddedWidth,
			hPaddedWidth * sizeof(uchar4), registeredImage + minx + miny * hWidth,
			hWidth * sizeof(uchar4), hWidthROI * sizeof(uchar4),
			hHeightROI, hipMemcpyHostToDevice));

	Uchar4ToFloat4(paddedRegisteredImageU4, paddedRegisteredImage, hPaddedWidth, hPaddedHeight);
}

__host__ void PadHostGrayImage(uchar4* registeredImage, float1 *paddedRegisteredGrayImage,
		int minx, int miny, int maxx, int maxy)
{
	hWidthROI = maxx - minx;
	hHeightROI = maxy - miny;

	int toaddxx = 0, toaddxy = 0, toaddyx = 0, toaddyy = 0;

	if (avSizeX) { toaddxx = hWidthROI * marginX / avSizeX; toaddxy = hHeightROI * marginY / avSizeX; }
	if (avSizeY) { toaddyx = hWidthROI * marginX / avSizeY; toaddyy = hHeightROI * marginY / avSizeY; }

	hPaddingSizeX = max(toaddxx, toaddyx); hPaddingSizeY = max(toaddxy, toaddyy);

	hPaddedWidth = hWidthROI + hPaddingSizeX*2;
	hPaddedHeight = hHeightROI + hPaddingSizeY*2;

	checkCudaErrors(hipMemset(paddedRegisteredImageU4, 0, sizeof(uchar4) * hPaddedWidth * hPaddedHeight));

	checkCudaErrors(hipMemcpy2D(paddedRegisteredImageU4 + hPaddingSizeX + hPaddingSizeY * hPaddedWidth,
			hPaddedWidth * sizeof(uchar4), registeredImage + minx + miny * hWidth,
			hWidth * sizeof(uchar4), hWidthROI * sizeof(uchar4),
			hHeightROI, hipMemcpyHostToDevice));
	
	checkCudaErrors(hipMemcpy(paddedRegisteredGrayImage, paddedRegisteredImageU4, hPaddedWidth * hPaddedHeight *
		sizeof(float1), hipMemcpyDeviceToDevice));
}
