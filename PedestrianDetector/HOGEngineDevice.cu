#include "hip/hip_runtime.h"
#include "HOGEngineDevice.h"
#include "HOGUtils.h"
#include "HOGConvolution.h"
#include "HOGHistogram.h"
#include "HOGSVMSlider.h"
#include "HOGScale.h"
#include "HOGPadding.h"

int hWidth, hHeight;
int hWidthROI, hHeightROI;
int hPaddedWidth, hPaddedHeight;
int rPaddedWidth, rPaddedHeight;

int minX, minY, maxX, maxY;

int hNoHistogramBins, rNoHistogramBins;

int hPaddingSizeX, hPaddingSizeY;
int hCellSizeX, hCellSizeY, hBlockSizeX, hBlockSizeY, hWindowSizeX, hWindowSizeY;
int hNoOfCellsX, hNoOfCellsY, hNoOfBlocksX, hNoOfBlocksY;
int rNoOfCellsX, rNoOfCellsY, rNoOfBlocksX, rNoOfBlocksY;

int hNumberOfBlockPerWindowX, hNumberOfBlockPerWindowY;
int hNumberOfWindowsX, hNumberOfWindowsY;
int rNumberOfWindowsX, rNumberOfWindowsY;

float4 *paddedRegisteredImage;
float1 *paddedRegisteredGrayImage;

float1 *resizedPaddedImageF1;
float4 *resizedPaddedImageF4;

float2 *colorGradientsF2;

float1 *blockHistograms;
float1 *cellHistograms;

float1 *svmScores;

bool hUseGrayscale;

// uchar1* outputTest1;
uchar4* outputTest4;

float* hResult;

float scaleRatio;
float startScale;
float endScale;
int scaleCount;

int avSizeX, avSizeY, marginX, marginY;

extern uchar4* paddedRegisteredImageU4;

float1 *deviceImage;

__host__ void InitHOG(int width, int height,
					  int _avSizeX, int _avSizeY,
					  int _marginX, int _marginY,
					  int cellSizeX, int cellSizeY,
					  int blockSizeX, int blockSizeY,
					  int windowSizeX, int windowSizeY,
					  int noOfHistogramBins, float wtscale,
					  float svmBias, float* svmWeights, int svmWeightsCount,
					  bool useGrayscale)
{
	hipSetDevice( gpuGetMaxGflopsDeviceId() );

	int i;
	int toaddxx = 0, toaddxy = 0, toaddyx = 0, toaddyy = 0;

	hWidth = width; hHeight = height;
	avSizeX = _avSizeX; avSizeY = _avSizeY; marginX = _marginX; marginY = _marginY;

	if (avSizeX) { toaddxx = hWidth * marginX / avSizeX; toaddxy = hHeight * marginY / avSizeX; }
	if (avSizeY) { toaddyx = hWidth * marginX / avSizeY; toaddyy = hHeight * marginY / avSizeY; }

	hPaddingSizeX = max(toaddxx, toaddyx); hPaddingSizeY = max(toaddxy, toaddyy);

	hPaddedWidth = hWidth + hPaddingSizeX*2;
	hPaddedHeight = hHeight + hPaddingSizeY*2;

	hUseGrayscale = useGrayscale;

	hNoHistogramBins = noOfHistogramBins;
	hCellSizeX = cellSizeX; hCellSizeY = cellSizeY; hBlockSizeX = blockSizeX; hBlockSizeY = blockSizeY;
	hWindowSizeX = windowSizeX; hWindowSizeY = windowSizeY;

	hNoOfCellsX = hPaddedWidth / cellSizeX;
	hNoOfCellsY = hPaddedHeight / cellSizeY;

	hNoOfBlocksX = hNoOfCellsX - blockSizeX + 1;
	hNoOfBlocksY = hNoOfCellsY - blockSizeY + 1;

	hNumberOfBlockPerWindowX = (windowSizeX - cellSizeX * blockSizeX) / cellSizeX + 1;
	hNumberOfBlockPerWindowY = (windowSizeY - cellSizeY * blockSizeY) / cellSizeY + 1;

	hNumberOfWindowsX = 0;
	for (i=0; i<hNumberOfBlockPerWindowX; i++) hNumberOfWindowsX += (hNoOfBlocksX-i)/hNumberOfBlockPerWindowX;

	hNumberOfWindowsY = 0;
	for (i=0; i<hNumberOfBlockPerWindowY; i++) hNumberOfWindowsY += (hNoOfBlocksY-i)/hNumberOfBlockPerWindowY;

	scaleRatio = 1.05f;
	startScale = 1.0f;
	endScale = min(hPaddedWidth / (float) hWindowSizeX, hPaddedHeight / (float) hWindowSizeY);
	scaleCount = (int)floor(logf(endScale/startScale)/logf(scaleRatio)) + 1;

	checkCudaErrors(hipMalloc((void**) &paddedRegisteredImage, sizeof(float4) * hPaddedWidth * hPaddedHeight));
	
	checkCudaErrors(hipMalloc((void**) &paddedRegisteredGrayImage, sizeof(float1) * hPaddedWidth * hPaddedHeight));

	if (useGrayscale)
		checkCudaErrors(hipMalloc((void**) &resizedPaddedImageF1, sizeof(float1) * hPaddedWidth * hPaddedHeight));
	else
		checkCudaErrors(hipMalloc((void**) &resizedPaddedImageF4, sizeof(float4) * hPaddedWidth * hPaddedHeight));

	checkCudaErrors(hipMalloc((void**) &colorGradientsF2, sizeof(float2) * hPaddedWidth * hPaddedHeight));
	checkCudaErrors(hipMalloc((void**) &blockHistograms, sizeof(float1) * hNoOfBlocksX * hNoOfBlocksY * cellSizeX * cellSizeY * hNoHistogramBins));
	checkCudaErrors(hipMalloc((void**) &cellHistograms, sizeof(float1) * hNoOfCellsX * hNoOfCellsY * hNoHistogramBins));

	checkCudaErrors(hipMalloc((void**) &svmScores, sizeof(float1) * hNumberOfWindowsX * hNumberOfWindowsY * scaleCount));

	InitConvolution(hPaddedWidth, hPaddedHeight, useGrayscale);
	InitHistograms(cellSizeX, cellSizeY, blockSizeX, blockSizeY, noOfHistogramBins, wtscale);
	InitSVM(svmBias, svmWeights, svmWeightsCount);
	InitScale(hPaddedWidth, hPaddedHeight);
	InitPadding(hPaddedWidth, hPaddedHeight);

	rPaddedWidth = hPaddedWidth;
	rPaddedHeight = hPaddedHeight;

	// if (useGrayscale)
		// checkCudaErrors(hipMalloc((void**) &outputTest1, sizeof(uchar1) * hPaddedWidth * hPaddedHeight));
	// else
		checkCudaErrors(hipMalloc((void**) &outputTest4, sizeof(uchar4) * hPaddedWidth * hPaddedHeight));

	checkCudaErrors(hipHostMalloc((void**)&hResult, sizeof(float) * hNumberOfWindowsX * hNumberOfWindowsY * scaleCount));
}

__host__ void CloseHOG()
{
	checkCudaErrors(hipFree(paddedRegisteredImage));
	
	checkCudaErrors(hipFree(paddedRegisteredGrayImage));

	if (hUseGrayscale)
		checkCudaErrors(hipFree(resizedPaddedImageF1));
	else
		checkCudaErrors(hipFree(resizedPaddedImageF4));

	checkCudaErrors(hipFree(colorGradientsF2));
	checkCudaErrors(hipFree(blockHistograms));
	checkCudaErrors(hipFree(cellHistograms));

	checkCudaErrors(hipFree(svmScores));

	CloseConvolution();
	CloseHistogram();
	CloseSVM();
	CloseScale();
	ClosePadding();

	// if (hUseGrayscale)
		// checkCudaErrors(hipFree(outputTest1));
	// else
		checkCudaErrors(hipFree(outputTest4));

	checkCudaErrors(hipHostFree(hResult));

	hipDeviceReset();
}

__host__ void BeginHOGProcessing(unsigned char* hostImage, int minx, int miny, int maxx, int maxy, float minScale, float maxScale)
{
	int i;
	minX = minx; minY = miny; maxX = maxx; maxY = maxy;
	
	if (hUseGrayscale) {
		PadHostGrayImage((uchar4*)hostImage, paddedRegisteredGrayImage, minX, minY, maxX, maxY);
	} else {
		PadHostImage((uchar4*)hostImage, paddedRegisteredImage, minX, minY, maxX, maxY);
	}
		
	rPaddedWidth = hPaddedWidth; rPaddedHeight = hPaddedHeight;
	scaleRatio = 1.05f;
	startScale = (minScale < 0.0f) ? 1.0f : minScale;
	endScale = (maxScale < 0.0f) ? min(hPaddedWidth / (float) hWindowSizeX, hPaddedHeight / (float) hWindowSizeY) : maxScale;
	scaleCount = (int)floor(logf(endScale/startScale)/logf(scaleRatio)) + 1;

	float currentScale = startScale;

	ResetSVMScores(svmScores);

	for (i=0; i<scaleCount; i++)
	{
		if (hUseGrayscale) {
			DownscaleGrayImage(0, scaleCount, i, currentScale, hUseGrayscale, paddedRegisteredGrayImage, resizedPaddedImageF1);
		} else {
			DownscaleImage(0, scaleCount, i, currentScale, hUseGrayscale, paddedRegisteredImage, resizedPaddedImageF1, resizedPaddedImageF4);
		}
		
		SetConvolutionSize(rPaddedWidth, rPaddedHeight);

		if (hUseGrayscale) ComputeColorGradients1to2(resizedPaddedImageF1, colorGradientsF2);
		else ComputeColorGradients4to2(resizedPaddedImageF4, colorGradientsF2);

		ComputeBlockHistogramsWithGauss(colorGradientsF2, blockHistograms, hNoHistogramBins,
			hCellSizeX, hCellSizeY, hBlockSizeX, hBlockSizeY, hWindowSizeX, hWindowSizeY,  rPaddedWidth, rPaddedHeight);

		NormalizeBlockHistograms(blockHistograms, hNoHistogramBins, hCellSizeX, hCellSizeY, hBlockSizeX, hBlockSizeY, rPaddedWidth, rPaddedHeight);

		LinearSVMEvaluation(svmScores, blockHistograms, hNoHistogramBins, hWindowSizeX, hWindowSizeY, hCellSizeX, hCellSizeY,
			hBlockSizeX, hBlockSizeY, rNoOfBlocksX, rNoOfBlocksY, i, rPaddedWidth, rPaddedHeight);

		currentScale *= scaleRatio;
	}
}

__host__ float* EndHOGProcessing()
{
	hipDeviceSynchronize();
	checkCudaErrors(hipMemcpy(hResult, svmScores, sizeof(float) * scaleCount * hNumberOfWindowsX * hNumberOfWindowsY, hipMemcpyDeviceToHost));
	return hResult;
}

__host__ void CalculateHOGDescriptor(float *hostImage, int width, int height, int noOfHistogramBins,
                                     int windowSizeX, int windowSizeY, int cellSizeX, int cellSizeY,
                                     int blockSizeX, int blockSizeY, float *hostDesc, float *gradient)
{
	hipSetDevice(gpuGetMaxGflopsDeviceId());
	
	int noOfBlocksX;
	int noOfBlocksY;
	float1 *deviceImage;
	float2 *colorGradientsF2;
	float1 *blockHistograms;
	int y;
	
	noOfBlocksX = width / cellSizeX - blockSizeX + 1;
	noOfBlocksY = height / cellSizeY - blockSizeY + 1;

	checkCudaErrors(hipMalloc((void**)&deviceImage, sizeof(float1) * width * height));

	checkCudaErrors(hipMalloc((void**)&colorGradientsF2, sizeof(float2) * width * height));

	checkCudaErrors(hipMalloc((void**)&blockHistograms, sizeof(float1) * noOfBlocksX * noOfBlocksY * blockSizeX * blockSizeY * noOfHistogramBins));
	
	checkCudaErrors(hipMalloc((void**) &outputTest4, sizeof(uchar4) * width * height));
	
	InitConvolution(width, height, true);
	InitHistograms(cellSizeX, cellSizeY, blockSizeX, blockSizeY, noOfHistogramBins, 2.0);
	
	checkCudaErrors(hipMemcpy2D(deviceImage, sizeof(float1) * width, hostImage, sizeof(float1) * width, sizeof(float1) * width, height, hipMemcpyHostToDevice));

	SetConvolutionSize(width, height);
	
	ComputeColorGradients1to2(deviceImage, colorGradientsF2);

	checkCudaErrors(hipMemcpy(colorGradientsF2, colorGradientsF2 + width, sizeof(float2) * width, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(colorGradientsF2 + width * (height - 1), colorGradientsF2 + width * (height - 2), sizeof(float2) * width, hipMemcpyDeviceToDevice));
	
	for (y = 0; y < height; y++) {
		checkCudaErrors(hipMemcpy(colorGradientsF2 + y * width, colorGradientsF2 + y * width + 1, sizeof(float2), hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(colorGradientsF2 + y * width + width - 1, colorGradientsF2 + y * width + width - 2, sizeof(float2), hipMemcpyDeviceToDevice));
	}
	
	ComputeBlockHistogramsWithGauss(colorGradientsF2, blockHistograms, noOfHistogramBins, cellSizeX, cellSizeY, blockSizeX, blockSizeY, windowSizeX, windowSizeY, width, height);

	NormalizeBlockHistograms(blockHistograms, noOfHistogramBins, cellSizeX, cellSizeY, blockSizeX, blockSizeY, width, height);

	Float2ToUchar4(colorGradientsF2, outputTest4, width, height, 0);
	checkCudaErrors(hipMemcpy2D(gradient, width * sizeof(float1), outputTest4, width * sizeof(float1), width * sizeof(float1), height, hipMemcpyDeviceToHost));
			
	checkCudaErrors(hipMemcpy(hostDesc, blockHistograms, sizeof(float1) * noOfBlocksX * noOfBlocksY * blockSizeX * blockSizeY * noOfHistogramBins, hipMemcpyDeviceToHost));
	
	checkCudaErrors(hipFree(deviceImage));
	checkCudaErrors(hipFree(colorGradientsF2));
	checkCudaErrors(hipFree(blockHistograms));
	checkCudaErrors(hipFree(outputTest4));
	
	CloseConvolution();
	CloseHistogram();
	
	hipDeviceReset();

	hipDeviceSynchronize();
}

__host__ void InitHOGDescriptorCalculator(int width, int height, int noOfHistogramBins,
                                          int windowSizeX, int windowSizeY, int cellSizeX, int cellSizeY,
                                          int blockSizeX, int blockSizeY, float wtscale)
{
	hWidth = width;
	hHeight = height;
	hNoHistogramBins = noOfHistogramBins;
	hWindowSizeX = windowSizeX;
	hWindowSizeY = windowSizeY;
	hCellSizeX = cellSizeX;
	hCellSizeY = cellSizeY;
	hBlockSizeX = blockSizeX;
	hBlockSizeY = blockSizeY;
	
	hNoOfBlocksX = width / cellSizeX - blockSizeX + 1;
	hNoOfBlocksY = height / cellSizeY - blockSizeY + 1;
	
	checkCudaErrors(hipMalloc((void**)&deviceImage, sizeof(float1) * width * height));

	checkCudaErrors(hipMalloc((void**)&colorGradientsF2, sizeof(float2) * width * height));

	checkCudaErrors(hipMalloc((void**)&blockHistograms, sizeof(float1) * hNoOfBlocksX * hNoOfBlocksY * blockSizeX * blockSizeY * noOfHistogramBins));
	
	checkCudaErrors(hipMalloc((void**) &outputTest4, sizeof(uchar4) * width * height));
	
	InitConvolution(width, height, true);
	InitHistograms(cellSizeX, cellSizeY, blockSizeX, blockSizeY, noOfHistogramBins, wtscale);
}

__host__ void HOGDescriptorCalculator(float *hostImage, float *hostDesc)
{
	int y;
	checkCudaErrors(hipMemcpy2D(deviceImage, sizeof(float1) * hWidth, hostImage, sizeof(float1) * hWidth, sizeof(float1) * hWidth, hHeight, hipMemcpyHostToDevice));
	
	SetConvolutionSize(hWidth, hHeight);
	
	ComputeColorGradients1to2(deviceImage, colorGradientsF2);

	checkCudaErrors(hipMemcpy(colorGradientsF2, colorGradientsF2 + hWidth, sizeof(float2) * hWidth, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipMemcpy(colorGradientsF2 + hWidth * (hHeight - 1), colorGradientsF2 + hWidth * (hHeight - 2), sizeof(float2) * hWidth, hipMemcpyDeviceToDevice));
	
	for (y = 0; y < hHeight; y++) {
		checkCudaErrors(hipMemcpy(colorGradientsF2 + y * hWidth, colorGradientsF2 + y * hWidth + 1, sizeof(float2), hipMemcpyDeviceToDevice));
		checkCudaErrors(hipMemcpy(colorGradientsF2 + y * hWidth + hWidth - 1, colorGradientsF2 + y * hWidth + hWidth - 2, sizeof(float2), hipMemcpyDeviceToDevice));
	}
	
	ComputeBlockHistogramsWithGauss(colorGradientsF2, blockHistograms, hNoHistogramBins, hCellSizeX, hCellSizeY, hBlockSizeX, hBlockSizeY, hWindowSizeX, hWindowSizeY, hWidth, hHeight);

	NormalizeBlockHistograms(blockHistograms, hNoHistogramBins, hCellSizeX, hCellSizeY, hBlockSizeX, hBlockSizeY, hWidth, hHeight);

	checkCudaErrors(hipMemcpy(hostDesc, blockHistograms, sizeof(float1) * hNoOfBlocksX * hNoOfBlocksY * hBlockSizeX * hBlockSizeY * hNoHistogramBins, hipMemcpyDeviceToHost));
}

__host__ void HOGDescriptorCalculatorOG(float *hostImage, float *hostDesc, float *gradient)
{
	HOGDescriptorCalculator(hostImage, hostDesc);
	
	Float2ToUchar4(colorGradientsF2, outputTest4, hWidth, hHeight, 0);
	checkCudaErrors(hipMemcpy2D(gradient, hWidth * sizeof(float1), outputTest4, hWidth * sizeof(float1), hWidth * sizeof(float1), hHeight, hipMemcpyDeviceToHost));
}

__host__ void FreeHOGDescriptorCalculator()
{
	checkCudaErrors(hipFree(deviceImage));
	checkCudaErrors(hipFree(colorGradientsF2));
	checkCudaErrors(hipFree(blockHistograms));
	checkCudaErrors(hipFree(outputTest4));
	
	CloseConvolution();
	CloseHistogram();
	
	hipDeviceReset();

	hipDeviceSynchronize();
}

__host__ void GetProcessedImage(unsigned char* hostImage, int imageType)
{
		switch (imageType)
		{
		case 0:
			Float4ToUchar4(resizedPaddedImageF4, outputTest4, rPaddedWidth, rPaddedHeight);
			break;
		case 1:
			Float2ToUchar4(colorGradientsF2, outputTest4, rPaddedWidth, rPaddedHeight, 0);
			break;
		case 2:
			Float2ToUchar4(colorGradientsF2, outputTest4, rPaddedWidth, rPaddedHeight, 1);
			break;
		case 3:
			checkCudaErrors(hipMemcpy(hostImage, paddedRegisteredImageU4, sizeof(uchar4) * hPaddedWidth * hPaddedHeight, hipMemcpyDeviceToHost));
			return;
		case 4:
			checkCudaErrors(hipMemcpy2D(((uchar4*)hostImage) + minX + minY * hWidth, hWidth * sizeof(uchar4), 
				paddedRegisteredImageU4 + hPaddingSizeX + hPaddingSizeY * hPaddedWidth, hPaddedWidth * sizeof(uchar4),
				hWidthROI * sizeof(uchar4), hHeightROI, hipMemcpyDeviceToHost));
			return;
		}

		checkCudaErrors(hipMemcpy2D(hostImage, hPaddedWidth * sizeof(uchar4), outputTest4, rPaddedWidth * sizeof(uchar4),
			rPaddedWidth * sizeof(uchar4), rPaddedHeight, hipMemcpyDeviceToHost));

	//checkCudaErrors(hipMemcpy(hostImage, paddedRegisteredImage, sizeof(uchar4) * hPaddedWidth * hPaddedHeight, hipMemcpyDeviceToHost));
}

__host__ void GetHOGDescriptor(float *hostDesc)
{
	checkCudaErrors(hipMemcpy(hostDesc, blockHistograms, sizeof(float1) * hNoOfBlocksX * hNoOfBlocksY * hCellSizeX * hCellSizeY * hNoHistogramBins, hipMemcpyDeviceToHost));
}

__host__ void GetHOGParameters(float *cStartScale, float *cEndScale, float *cScaleRatio, int *cScaleCount,
							   int *cPaddingSizeX, int *cPaddingSizeY, int *cPaddedWidth, int *cPaddedHeight,
							   int *cNoOfCellsX, int *cNoOfCellsY, int *cNoOfBlocksX, int *cNoOfBlocksY,
							   int *cNumberOfWindowsX, int *cNumberOfWindowsY,
							   int *cNumberOfBlockPerWindowX, int *cNumberOfBlockPerWindowY)
{
	*cStartScale = startScale;
	*cEndScale = endScale;
	*cScaleRatio = scaleRatio;
	*cScaleCount = scaleCount;
	*cPaddingSizeX = hPaddingSizeX;
	*cPaddingSizeY = hPaddingSizeY;
	*cPaddedWidth = hPaddedWidth;
	*cPaddedHeight = hPaddedHeight;
	*cNoOfCellsX = hNoOfCellsX;
	*cNoOfCellsY = hNoOfCellsY;
	*cNoOfBlocksX = hNoOfBlocksX;
	*cNoOfBlocksY = hNoOfBlocksY;
	*cNumberOfWindowsX = hNumberOfWindowsX;
	*cNumberOfWindowsY = hNumberOfWindowsY;
	*cNumberOfBlockPerWindowX = hNumberOfBlockPerWindowX;
	*cNumberOfBlockPerWindowY = hNumberOfBlockPerWindowY;
}

hipArray *imageArray2 = 0;
texture<float4, 2, hipReadModeElementType> tex2;
hipChannelFormatDesc channelDescDownscale2;

__global__ void resizeFastBicubic3(float4 *outputFloat, float4* paddedRegisteredImage, int width, int height, float scale)
{
	int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	int i = __umul24(y, width) + x;

	float u = x*scale;
	float v = y*scale;

	if (x < width && y < height)
	{
		float4 cF;

		if (scale == 1.0f)
			cF = paddedRegisteredImage[x + y * width];
		else
			cF = tex2D(tex2, u, v);

		outputFloat[i] = cF;
	}
}

__host__ void DownscaleImage2(float scale, float4* paddedRegisteredImage,
							  float4* resizedPaddedImageF4, int width, int height,
							  int &rPaddedWidth, int &rPaddedHeight)
{
	dim3 hThreadSize, hBlockSize;

	hThreadSize = dim3(THREAD_SIZE_W, THREAD_SIZE_H);

	rPaddedWidth = iDivUpF(width, scale);
	rPaddedHeight = iDivUpF(height, scale);

	hBlockSize = dim3(iDivUp(rPaddedWidth, hThreadSize.x), iDivUp(rPaddedHeight, hThreadSize.y));

	checkCudaErrors(hipMemcpyToArray(imageArray2, 0, 0, paddedRegisteredImage, sizeof(float4) * width * height, hipMemcpyDeviceToDevice));
	checkCudaErrors(hipBindTextureToArray(tex2, imageArray2, channelDescDownscale2));

	checkCudaErrors(hipMemset(resizedPaddedImageF4, 0, width * height * sizeof(float4)));
	resizeFastBicubic3<<<hBlockSize, hThreadSize>>>((float4*)resizedPaddedImageF4, (float4*)paddedRegisteredImage, rPaddedWidth, rPaddedHeight, scale);

	checkCudaErrors(hipUnbindTexture(tex2));
}

__host__ float3* CUDAImageRescale(float3* src, int width, int height, int &rWidth, int &rHeight, float scale)
{
	int i, j, offsetC, offsetL;

	float4* srcH; float4* srcD;
	float4* dstD; float4* dstH;
	float3 val3; float4 val4;

	channelDescDownscale2 = hipCreateChannelDesc<float4>();
	tex2.filterMode = hipFilterModeLinear; tex2.normalized = false;

	hipMalloc((void**)&srcD, sizeof(float4) * width * height);
	hipMalloc((void**)&dstD, sizeof(float4) * width * height);
	hipHostMalloc((void**)&srcH, sizeof(float4) * width * height);
	hipHostMalloc((void**)&dstH, sizeof(float4) * width * height);
	checkCudaErrors(hipMallocArray(&imageArray2, &channelDescDownscale2, width, height) );

	for (i=0; i<width; i++)
	{
		for (j=0; j<height; j++)
		{
			offsetC = j + i * height;
			offsetL = j * width + i;

			val3 = src[offsetC];

			srcH[offsetL].x = val3.x;
			srcH[offsetL].y = val3.y;
			srcH[offsetL].z = val3.z;
		}
	}
	hipMemcpy(srcD, srcH, sizeof(float4) * width * height, hipMemcpyHostToDevice);

	DownscaleImage2(scale, srcD, dstD, width, height, rWidth, rHeight);

	hipMemcpy(dstH, dstD, sizeof(float4) * rWidth * rHeight, hipMemcpyDeviceToHost);

	float3* dst = (float3*) malloc (rWidth * rHeight * sizeof(float3));
	for (i=0; i<rWidth; i++)
	{
		for (j=0; j<rHeight; j++)
		{
			offsetC = j + i * rHeight;
			offsetL = j * rWidth + i;

			val4 = dstH[offsetL];

			dst[offsetC].x = val4.x;
			dst[offsetC].y = val4.y;
			dst[offsetC].z = val4.z;
		}
	}

	checkCudaErrors(hipFreeArray(imageArray2));
	hipFree(srcD);
	hipFree(dstD);
	hipHostFree(srcH);
	hipHostFree(dstH);

	return dst;
}
