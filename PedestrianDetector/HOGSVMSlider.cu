#include "hip/hip_runtime.h"
#include "HOGSVMSlider.h"
#include "HOGUtils.h"

texture<float, 1, hipReadModeElementType> texSVM;
hipArray *svmArray = 0;

hipChannelFormatDesc channelDescSVM;

extern int scaleCount;
extern int hNumberOfWindowsX, hNumberOfWindowsY;
extern int hNumberOfBlockPerWindowX, hNumberOfBlockPerWindowY;
extern int rNumberOfWindowsX, rNumberOfWindowsY;

extern __shared__ float1 allSharedF1[];

float svmBias;

__host__ void InitSVM(float _svmBias, float* svmWeights, int svmWeightsCount)
{
	channelDescSVM = hipCreateChannelDesc<float>();
	checkCudaErrors(hipMallocArray(&svmArray, &channelDescSVM, svmWeightsCount, 1));
	checkCudaErrors(hipMemcpyToArray(svmArray, 0, 0, svmWeights, svmWeightsCount * sizeof(float), hipMemcpyHostToDevice));
	svmBias = _svmBias;
}

__host__ void CloseSVM()
{
	checkCudaErrors(hipFreeArray(svmArray));
}

__global__ void linearSVMEvaluation(float1* svmScores, float svmBias,
									float1* blockHistograms, int noHistogramBins,
									int windowSizeX, int windowSizeY, int hogBlockCountX, int hogBlockCountY,
									int cellSizeX, int cellSizeY,
									int numberOfBlockPerWindowX, int numberOfBlockPerWindowY,
									int blockSizeX, int blockSizeY,
									int alignedBlockDimX,
									int scaleId, int scaleCount,
									int hNumberOfWindowsX, int hNumberOfWindowsY,
									int width, int height)
{
	int i;
	int texPos;
	float1 localValue;
	float texValue;

	float1* smem = (float1*) allSharedF1;

	int gmemPosWindow, gmemPosInWindow, gmemPosInWindowDown, smemLocalPos, smemTargetPos;
	int gmemStride = hogBlockCountX * noHistogramBins * blockSizeX;
	
	// printf("blockHistograms %p\n", blockHistograms);
	
	// printf("gmemStride %d\n", gmemStride); // = 54

	gmemPosWindow = blockIdx.x * noHistogramBins * blockSizeX + blockIdx.y * blockSizeY * gmemStride;
	
	// if (blockIdx.x == 1 && blockIdx.y == 1)
	// printf("blockIdx.x %d, blockIdx.y %d\n", blockIdx.x, blockIdx.y);
	
	// printf("gmemPosWindow %d\n", gmemPosWindow); // = 0
	
	gmemPosInWindow = gmemPosWindow + threadIdx.x;
	
	// printf("threadIdx.x %d\n", threadIdx.x);
	
	// printf("gmemPosInWindow %d\n", gmemPosInWindow); // = 0 ~ 53
	
	smemLocalPos = threadIdx.x;

	int val1 = (blockSizeY * blockSizeX * noHistogramBins) * numberOfBlockPerWindowY;
	int val2 = blockSizeX * noHistogramBins; // 18
	localValue.x = 0;

	if (blockIdx.x == 10 && blockIdx.y == 8)
	{
		int asasasa;
		asasasa = 0;
		asasasa++;
	}

	for (i = 0; i<blockSizeY * numberOfBlockPerWindowY; i++)
	{
		gmemPosInWindowDown = gmemPosInWindow + i * gmemStride;
		texPos = threadIdx.x % val2 + i * val2 + threadIdx.x / val2 * val1;
		
		// if (blockIdx.x == 1 && blockIdx.y == 1 && threadIdx.x == 53)
		// printf("threadIdx.x %d, row %d, texPos %d\n", threadIdx.x, i, texPos);
		
		texValue =  tex1D(texSVM, texPos);
		localValue.x += blockHistograms[gmemPosInWindowDown].x * texValue;
	}

	smem[smemLocalPos] = localValue;

	__syncthreads();

	for(unsigned int s = alignedBlockDimX >> 1; s>0; s>>=1)
	{
		// printf("s %u\n", s);
		if (threadIdx.x < s && (threadIdx.x + s) < blockDim.x)
		{
			smemTargetPos = threadIdx.x + s;
			smem[smemLocalPos].x += smem[smemTargetPos].x;
			// printf("%d %u %d\n", smemLocalPos, s, smemTargetPos);
		}

		__syncthreads();
	}

	if (threadIdx.x == 0)
	{
		smem[smemLocalPos].x -= svmBias;
		svmScores[blockIdx.x + blockIdx.y * hNumberOfWindowsX + scaleId * hNumberOfWindowsX * hNumberOfWindowsY] = smem[smemLocalPos];
	}

	if (blockIdx.x == 10 && blockIdx.y == 8)
	{
		int asasasa;
		asasasa = 0;
		asasasa++;
	}
}

__host__ void ResetSVMScores(float1* svmScores)
{
	checkCudaErrors(hipMemset(svmScores, 0, sizeof(float) * scaleCount * hNumberOfWindowsX * hNumberOfWindowsY));
}

__host__ void LinearSVMEvaluation(float1* svmScores, float1* blockHistograms, int noHistogramBins,
								  int windowSizeX, int windowSizeY,
								  int cellSizeX, int cellSizeY, int blockSizeX, int blockSizeY,
								  int hogBlockCountX, int hogBlockCountY,
								  int scaleId, int width, int height)
{
	rNumberOfWindowsX = (width-windowSizeX)/cellSizeX + 1;
	rNumberOfWindowsY = (height-windowSizeY)/cellSizeY + 1;

	dim3 threadCount = dim3(noHistogramBins * blockSizeX * hNumberOfBlockPerWindowX);
	dim3 blockCount = dim3(rNumberOfWindowsX, rNumberOfWindowsY);

	int alignedBlockDimX = iClosestPowerOfTwo(noHistogramBins * blockSizeX * hNumberOfBlockPerWindowX);

	checkCudaErrors(hipBindTextureToArray(texSVM, svmArray, channelDescSVM));

	linearSVMEvaluation<<<blockCount, threadCount, noHistogramBins * blockSizeX * hNumberOfBlockPerWindowX * sizeof(float1)>>>
		(svmScores, svmBias, blockHistograms, noHistogramBins,
		windowSizeX, windowSizeY, hogBlockCountX, hogBlockCountY, cellSizeX, cellSizeY,
		hNumberOfBlockPerWindowX, hNumberOfBlockPerWindowY,
		blockSizeX, blockSizeY, alignedBlockDimX, scaleId, scaleCount,
		hNumberOfWindowsX, hNumberOfWindowsY, width, height);

	checkCudaErrors(hipUnbindTexture(texSVM));
}
