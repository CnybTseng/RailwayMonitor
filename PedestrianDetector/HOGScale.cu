#include "hip/hip_runtime.h"
#include "HOGScale.h"
#include "HOGUtils.h"

extern int rPaddedHeight;
extern int rPaddedWidth;
extern int hPaddedHeight;
extern int hPaddedWidth;
hipArray *imageArray = 0;
texture<float4, 2, hipReadModeElementType> tex;
texture<float1, 2, hipReadModeElementType> grayTex;
hipChannelFormatDesc channelDescDownscale;
hipChannelFormatDesc grayChannelDescDownscale;

bool isAlocated;

// w0, w1, w2, and w3 are the four cubic B-spline basis functions
__device__ float w0(float a) { return (1.0f/6.0f)*(a*(a*(-a + 3.0f) - 3.0f) + 1.0f); }
__device__ float w1(float a) { return (1.0f/6.0f)*(a*a*(3.0f*a - 6.0f) + 4.0f); }
__device__ float w2(float a) { return (1.0f/6.0f)*(a*(a*(-3.0f*a + 3.0f) + 3.0f) + 1.0f); }
__device__ float w3(float a) { return (1.0f/6.0f)*(a*a*a); }

// g0 and g1 are the two amplitude functions
__device__ float g0(float a) { return w0(a) + w1(a); }
__device__ float g1(float a) { return w2(a) + w3(a); }

// h0 and h1 are the two offset functions
__device__ float h0(float a) { return -1.0f + w1(a) / (w0(a) + w1(a)) + 0.5f; }
__device__ float h1(float a) { return 1.0f + w3(a) / (w2(a) + w3(a)) + 0.5f; }

__host__ void InitScale(int hPaddedWidth, int hPaddedHeight)
{
	channelDescDownscale = hipCreateChannelDesc<float4>();
	grayChannelDescDownscale = hipCreateChannelDesc<float1>();
	tex.filterMode = hipFilterModeLinear;
	tex.normalized = false;
	grayTex.filterMode = hipFilterModeLinear;
	grayTex.normalized = false;
	isAlocated = false;
}

__host__ void CloseScale()
{
	//if (isAlocated) checkCudaErrors(hipFreeArray(imageArray));
}

__host__ void DownscaleImage(int startScaleId, int endScaleId, int scaleId, float scale, 
							 bool useGrayscale, float4* paddedRegisteredImage,
							 float1* resizedPaddedImageF1, float4* resizedPaddedImageF4)
{
	dim3 hThreadSize, hBlockSize;

	hThreadSize = dim3(THREAD_SIZE_W, THREAD_SIZE_H);

	rPaddedWidth = iDivUpF(hPaddedWidth, scale);
	rPaddedHeight = iDivUpF(hPaddedHeight, scale);

	hBlockSize = dim3(iDivUp(rPaddedWidth, hThreadSize.x), iDivUp(rPaddedHeight, hThreadSize.y));

	if (scaleId == startScaleId)
	{
		if (isAlocated)
			checkCudaErrors(hipFreeArray(imageArray));
		checkCudaErrors(hipMallocArray(&imageArray, &channelDescDownscale, hPaddedWidth, hPaddedHeight) );
		checkCudaErrors(hipMemcpyToArray(imageArray, 0, 0, paddedRegisteredImage, sizeof(float4) * hPaddedWidth * hPaddedHeight, hipMemcpyDeviceToDevice));
		isAlocated = true;
	}

	checkCudaErrors(hipBindTextureToArray(tex, imageArray, channelDescDownscale));

	if (useGrayscale)
	{
		checkCudaErrors(hipMemset(resizedPaddedImageF1, 0, hPaddedWidth * hPaddedHeight * sizeof(float1)));
		resizeFastBicubic1<<<hBlockSize, hThreadSize>>>(resizedPaddedImageF1, paddedRegisteredImage, rPaddedWidth, rPaddedHeight, scale);
	}
	else
	{
		checkCudaErrors(hipMemset(resizedPaddedImageF4, 0, hPaddedWidth * hPaddedHeight * sizeof(float4)));
		resizeFastBicubic4<<<hBlockSize, hThreadSize>>>(resizedPaddedImageF4, paddedRegisteredImage, rPaddedWidth, rPaddedHeight, scale);
	}

	checkCudaErrors(hipUnbindTexture(tex));

	if (scaleId == endScaleId)
	{
		checkCudaErrors(hipFreeArray(imageArray));
		isAlocated = false;
	}
}

__host__ void DownscaleGrayImage(int startScaleId, int endScaleId, int scaleId, float scale, 
							 bool useGrayscale, float1* paddedRegisteredGrayImage,
							 float1* resizedPaddedImageF1)
{
	dim3 hThreadSize, hBlockSize;

	hThreadSize = dim3(THREAD_SIZE_W, THREAD_SIZE_H);

	rPaddedWidth = iDivUpF(hPaddedWidth, scale);
	rPaddedHeight = iDivUpF(hPaddedHeight, scale);

	hBlockSize = dim3(iDivUp(rPaddedWidth, hThreadSize.x), iDivUp(rPaddedHeight, hThreadSize.y));

	if (scaleId == startScaleId)
	{
		if (isAlocated)
			checkCudaErrors(hipFreeArray(imageArray));
		checkCudaErrors(hipMallocArray(&imageArray, &grayChannelDescDownscale, hPaddedWidth, hPaddedHeight) );
		checkCudaErrors(hipMemcpyToArray(imageArray, 0, 0, paddedRegisteredGrayImage, sizeof(float1) * hPaddedWidth * hPaddedHeight, hipMemcpyDeviceToDevice));
		isAlocated = true;
	}

	checkCudaErrors(hipBindTextureToArray(grayTex, imageArray, grayChannelDescDownscale));

	checkCudaErrors(hipMemset(resizedPaddedImageF1, 0, hPaddedWidth * hPaddedHeight * sizeof(float1)));
	resizeGrayFastBicubic1<<<hBlockSize, hThreadSize>>>(resizedPaddedImageF1, paddedRegisteredGrayImage, rPaddedWidth, rPaddedHeight, scale);

	checkCudaErrors(hipUnbindTexture(grayTex));

	if (scaleId == endScaleId)
	{
		checkCudaErrors(hipFreeArray(imageArray));
		isAlocated = false;
	}
}

__device__ float4 tex2DFastBicubic(const texture<float4, 2, hipReadModeElementType> texref, float x, float y)
{
	float4 r;
	float4 val0, val1, val2, val3;

	x -= 0.5f;
	y -= 0.5f;
	float px = floor(x);
	float py = floor(y);
	float fx = x - px;
	float fy = y - py;

	float g0x = g0(fx);
	float g1x = g1(fx);
	float h0x = h0(fx);
	float h1x = h1(fx);
	float h0y = h0(fy);
	float h1y = h1(fy);

	val0 = tex2D(texref, px + h0x, py + h0y);
	val1 = tex2D(texref, px + h1x, py + h0y);
	val2 = tex2D(texref, px + h0x, py + h1y);
	val3 = tex2D(texref, px + h1x, py + h1y);

	r.x = (g0(fy) * (g0x * val0.x + g1x * val1.x) + g1(fy) * (g0x * val2.x + g1x * val3.x));
	r.y = (g0(fy) * (g0x * val0.y + g1x * val1.y) + g1(fy) * (g0x * val2.y + g1x * val3.y));
	r.z = (g0(fy) * (g0x * val0.z + g1x * val1.z) + g1(fy) * (g0x * val2.z + g1x * val3.z));
	r.w = (g0(fy) * (g0x * val0.w + g1x * val1.w) + g1(fy) * (g0x * val2.w + g1x * val3.w));

	return r;
}

__global__ void resizeFastBicubic4(float4 *outputFloat, float4* paddedRegisteredImage, int width, int height, float scale)
{
	int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	int i = __umul24(y, width) + x;

	float u = x*scale;
	float v = y*scale;

	if (x < width && y < height)
	{
		float4 cF;

		if (scale == 1.0f)
		{
			cF = paddedRegisteredImage[x + y * width];
			cF.w = 0;
		}
		else
		{
			cF = tex2D(tex, u, v);
			cF.w = 0;
		}

		cF.x = sqrtf(cF.x); cF.y = sqrtf(cF.y); cF.z = sqrtf(cF.z); cF.w = 0;
		outputFloat[i] = cF;
	}
}

__global__ void resizeFastBicubic1(float1 *outputFloat, float4* paddedRegisteredImage, int width, int height, float scale)
{
	int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	int i = __umul24(y, width) + x;

	float u = x*scale;
	float v = y*scale;

	if (x < width && y < height)
	{
		float4 cF;

		if (scale == 1.0f)
		{
			cF = paddedRegisteredImage[x + y * width];
			cF.w = 0;
		}
		else
		{
			cF = tex2D(tex, u, v);
			cF.w = 0;
		}

		outputFloat[i].x = sqrtf(0.2989f * cF.x + 0.5870f * cF.y + 0.1140f * cF.z);
	}
}

__global__ void resizeGrayFastBicubic1(float1 *outputFloat, float1* paddedRegisteredGrayImage, int width, int height, float scale)
{
	int x = __umul24(blockIdx.x, blockDim.x) + threadIdx.x;
	int y = __umul24(blockIdx.y, blockDim.y) + threadIdx.y;
	int i = __umul24(y, width) + x;

	float u = x*scale;
	float v = y*scale;

	if (x < width && y < height)
	{
		float1 cF;

		if (scale == 1.0f)
		{
			cF = paddedRegisteredGrayImage[x + y * width];
		}
		else
		{
			cF = tex2D(grayTex, u, v);
		}

		outputFloat[i].x = cF.x;
	}
}
